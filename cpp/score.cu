#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "kernel_args.h"  // for intellisense
#include "score.h"

/**
 * Kernel to compute dot product of a single query vector relative to a batch of vectors.
 */
__global__ void multiplyAndSum(
    const float* batch,
    const float* query,
    float* dotResults,
    float* normResults,
    size_t batchSize,
    size_t vectorSize
) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < batchSize * vectorSize) {
        size_t batchIdx = idx / vectorSize;
        size_t vectorIdx = idx % vectorSize;
        atomicAdd(&dotResults[batchIdx], batch[idx] * query[vectorIdx]);
        atomicAdd(&normResults[batchIdx], batch[idx] * batch[idx]);
    }
}

__global__ void normalize(
    float* dotResults,
    const float* normResults,
    float queryNorm,
    size_t batchSize
) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < batchSize) {
        dotResults[idx] /= (sqrtf(normResults[idx]) * queryNorm);
    }
}


void cudaCosineSimilarity(
    const float* batch,
    const float* query,
    float* results,
    size_t batchSize,
    size_t vectorSize
) {
    float queryNorm = norm(query, vectorSize);

    // allocate cuda memory
    float* cudaBatch;
    float* cudaQuery;
    float* cudaDotResults;
    float* cudaNormResults;
    size_t memSize = batchSize * sizeof(float);
    hipMalloc(&cudaBatch, memSize * vectorSize);
    hipMalloc(&cudaQuery, memSize);
    hipMalloc(&cudaDotResults, memSize);
    hipMalloc(&cudaNormResults, memSize);
    hipMemcpy(cudaBatch, batch, memSize * vectorSize, hipMemcpyHostToDevice);
    hipMemcpy(cudaQuery, query, memSize, hipMemcpyHostToDevice);
    hipMemset(cudaDotResults, 0, memSize);
    hipMemset(cudaNormResults, 0, memSize);

    // run kernel
    int threads = 256;
    int blocks = ((int)(batchSize * vectorSize) + threads - 1) / threads;
    multiplyAndSum KERNEL_ARGS2(blocks, threads) (cudaBatch, cudaQuery, cudaDotResults, cudaNormResults, batchSize, vectorSize);
    hipDeviceSynchronize();

    blocks = ((int)batchSize + threads - 1) / threads;
    normalize KERNEL_ARGS2(blocks, threads) (cudaDotResults, cudaNormResults, queryNorm, batchSize);
    hipDeviceSynchronize();

    // copy results to cpu and free cuda memory
    hipMemcpy(results, cudaDotResults, memSize, hipMemcpyDeviceToHost);
    hipFree(cudaBatch);
    hipFree(cudaQuery);
    hipFree(cudaDotResults);
    hipFree(cudaNormResults);
}
